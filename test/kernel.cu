#include "hip/hip_runtime.h"
// Inner loop for prog_seed 3,000
DEV_INLINE void progPowLoop(const uint32_t loop,
    uint32_t mix[PROGPOW_REGS],
    const dag_t* g_dag,
    const uint32_t c_dag[PROGPOW_CACHE_WORDS],
    const bool hack_false,
    const uint32_t lane_id)
{
    dag_t data_dag;
    uint32_t offset, data;
    // global load
    offset = _SHFL(mix[0], loop & (PROGPOW_LANES - 1), PROGPOW_LANES);
    offset %= PROGPOW_DAG_ELEMENTS;
    offset = offset * PROGPOW_LANES + ((lane_id ^ loop) & (PROGPOW_LANES - 1));
    data_dag = g_dag[offset];
    // hack to prevent compiler from reordering LD and usage
    if (hack_false) __threadfence_block();
    // cache load 0
    offset = mix[12] & (PROGPOW_CACHE_WORDS - 1);
    data = c_dag[offset];
    mix[26] = ROTR32(mix[26], 17) ^ data;
    // random math 0
    data = mix[13] ^ mix[3];
    mix[9] = ROTL32(mix[9], 17) ^ data;
    // cache load 1
    offset = mix[1] & (PROGPOW_CACHE_WORDS - 1);
    data = c_dag[offset];
    mix[15] = ROTL32(mix[15], 15) ^ data;
    // random math 1
    data = mix[24] ^ mix[10];
    mix[16] = (mix[16] * 33) + data;
    // cache load 2
    offset = mix[29] & (PROGPOW_CACHE_WORDS - 1);
    data = c_dag[offset];
    mix[25] = (mix[25] ^ data) * 33;
    // random math 2
    data = ROTL32(mix[4], mix[12]);
    mix[12] = ROTR32(mix[12], 13) ^ data;
    // cache load 3
    offset = mix[6] & (PROGPOW_CACHE_WORDS - 1);
    data = c_dag[offset];
    mix[7] = ROTL32(mix[7], 8) ^ data;
    // random math 3
    data = mix[8] * mix[24];
    mix[31] = (mix[31] ^ data) * 33;
    // cache load 4
    offset = mix[11] & (PROGPOW_CACHE_WORDS - 1);
    data = c_dag[offset];
    mix[27] = ROTL32(mix[27], 2) ^ data;
    // random math 4
    data = popcount(mix[28]) + popcount(mix[17]);
    mix[5] = (mix[5] * 33) + data;
    // cache load 5
    offset = mix[18] & (PROGPOW_CACHE_WORDS - 1);
    data = c_dag[offset];
    mix[11] = ROTR32(mix[11], 28) ^ data;
    // random math 5
    data = mix[31] ^ mix[12];
    mix[17] = (mix[17] ^ data) * 33;
    // cache load 6
    offset = mix[8] & (PROGPOW_CACHE_WORDS - 1);
    data = c_dag[offset];
    mix[29] = ROTR32(mix[29], 10) ^ data;
    // random math 6
    data = popcount(mix[4]) + popcount(mix[12]);
    mix[10] = (mix[10] * 33) + data;
    // cache load 7
    offset = mix[14] & (PROGPOW_CACHE_WORDS - 1);
    data = c_dag[offset];
    mix[6] = (mix[6] ^ data) * 33;
    // random math 7
    data = min(mix[10], mix[20]);
    mix[24] = (mix[24] * 33) + data;
    // cache load 8
    offset = mix[17] & (PROGPOW_CACHE_WORDS - 1);
    data = c_dag[offset];
    mix[14] = (mix[14] ^ data) * 33;
    // random math 8
    data = mix[0] * mix[10];
    mix[19] = ROTR32(mix[19], 23) ^ data;
    // cache load 9
    offset = mix[9] & (PROGPOW_CACHE_WORDS - 1);
    data = c_dag[offset];
    mix[23] = (mix[23] * 33) + data;
    // random math 9
    data = min(mix[22], mix[28]);
    mix[1] = ROTR32(mix[1], 4) ^ data;
    // cache load 10
    offset = mix[0] & (PROGPOW_CACHE_WORDS - 1);
    data = c_dag[offset];
    mix[18] = (mix[18] ^ data) * 33;
    // random math 10
    data = ROTL32(mix[22], mix[9]);
    mix[21] = ROTR32(mix[21], 5) ^ data;
    // random math 11
    data = min(mix[26], mix[4]);
    mix[22] = (mix[22] * 33) + data;
    // random math 12
    data = min(mix[19], mix[30]);
    mix[8] = ROTL32(mix[8], 26) ^ data;
    // random math 13
    data = mix[12] ^ mix[24];
    mix[3] = ROTL32(mix[3], 30) ^ data;
    // random math 14
    data = min(mix[8], mix[13]);
    mix[28] = ROTL32(mix[28], 31) ^ data;
    // random math 15
    data = ROTL32(mix[12], mix[9]);
    mix[30] = ROTL32(mix[30], 31) ^ data;
    // random math 16
    data = ROTL32(mix[28], mix[27]);
    mix[2] = (mix[2] * 33) + data;
    // random math 17
    data = ROTL32(mix[30], mix[28]);
    mix[20] = ROTL32(mix[20], 12) ^ data;
    // consume global load data
    // hack to prevent compiler from reordering LD and usage
    if (hack_false) __threadfence_block();
    mix[0] = (mix[0] * 33) + data_dag.words[0];
    mix[4] = ROTL32(mix[4], 13) ^ data_dag.words[1];
    mix[13] = (mix[13] ^ data_dag.words[2]) * 33;
    mix[0] = ROTR32(mix[0], 12) ^ data_dag.words[3];
}
